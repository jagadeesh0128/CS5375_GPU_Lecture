
/*
 * _MATRIXMUL_GPU_CU_
 *
 * 2022 Mert SIDE
 *
 * CS5375 Computer Systems Organization and Architecture
 * Guest Lecture: GPU Programming
 *
 * Multiplying two matrices on the GPU
 *
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

// ------------------------------------------------------------------ GPUmatmul
//Implemented threads by using stride 
__global__
void GPUmatmul(int N, double *x, double *y, double *ans)
{
int indexx=threadIdx.x;
int indexy=threadIdx.y;
int stridex= blockDim.x;
int stridey= blockDim.y;
for (int i= indexx; i <N; i+=stridex)
{
   for (int j= indexy;j<N;j+=stridey)
    {  for (int k=0;k<N;k++)
        {
            ans[i*N+j]+=(x[i*N+k]*y[k*N+j]);
        }
    }
}
}



// ---------------------------------------------------------------------- check
bool check(int N, double *ans)
{
                                                           for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

// ----------------------------------------------------------------------- MAIN
int main(void)
{
  // size of matrix
  int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
  printf("Size of matrix (N) is %d by %d.\n", N, N);
  int iter = 3;
  clock_t t;

  // Martices
  double *x, *y, *ans;
   // Allocate memory accessible to both CPU and GPU
  hipMallocManaged(&x,sizeof(double)*N*N);
  hipMallocManaged(&y,sizeof(double)*N*N);
  hipMallocManaged(&ans,sizeof(double)*N*N);
    // ..........................................................................
  // initialize x,y and ans arrays on the host
  for (int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      x[i*N+j] = 5;
      y[i*N+j] = (i==j?1:0);
      ans[i*N+j] = (double)0.000000000000;
    }
  }

  // ..........................................................................
  double avg=0;
  std::cout<<"Starting unoptimized GPU computation"<<std::endl;
  // Run kernel on GPU
  for(int i = 0; i <= iter; i++) {
    t = clock();
    GPUmatmul<<<1,256>>>(N, x, y,ans);   //Changed threads to 256
    hipDeviceSynchronize();
    t = clock() - t;
    if(i) avg += t; //we will ignore the first run
    // printf ("It took GPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
  }
  
  avg /= iter;
  avg /= CLOCKS_PER_SEC;
  avg *= 1000;
  printf("It took %lf ms on avg.\n", avg);
  if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
  else std::cout<<"RUN NOT OK."<<std::endl;

  // ..........................................................................

  // Free memory
  // ...

  hipFree(x);
  hipFree(y);
  hipFree(ans);
  return 0;
}
/* EOF */
